#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <math.h>

extern "C" {
    #include "NeuralNetwork.h"
}

__global__ void sumExp(float *Z2, float *sum_exp){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int Z_x_dim = 60000;
    int Z_y_dim = 10;

    if(idx < Z_x_dim*Z_y_dim){
        atomicAdd(sum_exp, exp(Z2[idx]));
    }
}

__global__ void softMax(float *Z2, float *A2, float *sum_exp, 
    int Z_x_dim, int Z_y_dim){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < Z_x_dim*Z_y_dim){
        A2[idx] = exp(Z2[idx]) * *sum_exp;
    }
}

__global__ void reLUBack(float* Z1, float* dA1, int Z_x_dim, int Z_y_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < Z_x_dim * Z_y_dim) {
        if(Z1[idx] > 0){
            dA1[idx] *= Z1[idx];
        } else {
            dA1[idx] *= 0;
        }
    }
}

__global__ void reLUForward(float *Z1, float *A1, int Z_x_dim, int Z_y_dim){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;

    if(idx < Z_x_dim * Z_y_dim){
        if(Z1[idx] > 0){
            //A1[idx] = Z1[idx];
        } else {
            A1[0] = 0;
        }
    }
}

__global__ void linearForwardProp(float* A, float* Z, ParametersLinear *params, int *num_images, int *num_rows, int *num_cols,
    int Z_x_dim, int Z_y_dim, int W_x_dim, int W_y_dim){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float Z_value = 0;

    float *W = params->W;
    float *B = params->B; 

    float val = params->W[0];

    if(idx < Z_x_dim && idy < Z_y_dim){
        for(int t=0; t< W_y_dim; t++){
            Z_value += A[idx*Z_x_dim + t] * params->W[idy * W_y_dim + t] + params->B[idy];
        }
        
        Z[idx * Z_y_dim + idy] = Z_value;
    }
}

__global__ void linearBackProp(float *dZ2, float *dZ1, ParametersLinear *params, int Z_x_dim, int Z_y_dim, int W_y_dim){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float dZ1_value = 0.0f;

    if(idx < Z_x_dim && idy < Z_y_dim){
        for(int i=0; i<W_y_dim; i++){
            dZ1_value += params->W[i*W_y_dim + idx] * dZ2[i*W_y_dim + idy];
        }
        dZ2[idx*Z_y_dim+idy] = dZ1_value;
    }
}

__global__ void linearUpdateWeight(float* A, float* dZ, ParametersLinear *params, int W_x_dim, int W_y_dim, int A_x_dim, int A_y_dim, int dZ_x_dim){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float dW_value = 0.0f;

    float learning_rate = 0.01;

    if(idx < W_x_dim && idy < W_y_dim){
        for(int i=0; i<dZ_x_dim; i++){
            dW_value += dZ[idx*W_y_dim + i] * A[idy * A_y_dim + i];
        }

        params->W[idx * W_y_dim + idy] = params->W[idx * W_y_dim + idy] - learning_rate * dW_value/A_x_dim;
    }
}

__global__ void linearUpdateBias(float *dZ, ParametersLinear *params, int dZ_x_dim, int dZ_y_dim){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;

    float learning_rate = 0.01;

    if(idx < dZ_x_dim * dZ_y_dim){
        int dZ_x = idx / dZ_x_dim;
        int dZ_y = idx % dZ_x_dim;
        atomicAdd(&params->B[dZ_y], -learning_rate * (dZ[dZ_x * dZ_y_dim + dZ_y] / dZ_y_dim));
    }
}

void ForwardProp(float *d_X, ParametersLinear *d_params1, ParametersLinear *d_params2, int *d_numImages, int *d_numRows, 
    int *d_numCols, int *h_numImages, int *h_numRows, int *h_numCols,
    float *d_Z1, float *d_A1, float *d_Z2, float *d_A2){

    float *d_sum_exp; 
    hipMalloc((void**)&d_sum_exp, sizeof(float));

    int matrixSize = *h_numImages * 784;
    printf("Start Forward \n");
    hipError_t hipError_t;

    int batchSize = 1000;

    int numBatches = (*h_numImages) / batchSize;

    int batcharraysize = batchSize * 784;

    dim3 block_size(8,8);
    dim3 num_of_blocks((batchSize+block_size.x-1)/block_size.x,(*h_numRows * *h_numCols+block_size.y-1)/block_size.y);

    int singleDimblockSize = 1028;
    int singleDimnumBlocks = (batcharraysize + singleDimblockSize - 1) / singleDimblockSize;

    int singleDimnumBlocks2 = (batchSize * 10 + singleDimblockSize - 1) / singleDimblockSize;

    printf("Matrix Size %i \n", matrixSize);
    printf("Num block x threads %i \n", singleDimblockSize * singleDimnumBlocks);
    printf("Num Threads %i \n", singleDimnumBlocks);

    for (int i = 0; i < numBatches; i++) {
        printf("Batch %i Start \n", i);
        // Calculate the start and end indices for this batch
        int startIdx = i * batchSize;
        int endIdx = min(startIdx + batchSize, *h_numImages);

        // Calculate the size of this batch
        int batchMatrixSize = (endIdx - startIdx) * 784;

        /// First Linear Layer
        linearForwardProp<<<num_of_blocks, block_size>>>(d_X + startIdx * 784, d_Z1 + startIdx * 10, d_params1, d_numImages, d_numRows, d_numCols,
            batchSize, 10, 10 , 784);
        hipDeviceSynchronize();
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearForwardProp1): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        /// Relu Layer
        reLUForward<<<singleDimblockSize, singleDimnumBlocks>>>(d_Z1 + startIdx * 10, d_A1 + startIdx * 10, batchSize, 10);
        hipDeviceSynchronize();

        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (reLUForward): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        /// Second Linear Layer
        linearForwardProp<<<num_of_blocks, block_size>>>(d_A1 + startIdx * 10, d_Z2 + startIdx * 10, d_params2, d_numImages, d_numRows, d_numCols, 
            batchSize, 10, 10, 10);
        hipDeviceSynchronize();

        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearForwardProp2): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }

        /// Sum the exponetial 
        sumExp<<<singleDimblockSize, singleDimnumBlocks2>>>(d_Z2 + startIdx * 10, d_sum_exp);
        hipDeviceSynchronize();

        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (sumExp): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        /// Conduct the SoftMax
        softMax<<<singleDimblockSize, singleDimnumBlocks2>>>(d_Z2 + startIdx * 10, d_A2 + startIdx * 10, d_sum_exp, 
        1000, 10);
        hipDeviceSynchronize();

        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (softMax): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
    }
}

__global__ void startBackProp(float *d_Z2, float *d_A2, unsigned char *d_one_hot_Y, int Z_x_dim, int Z_y_dim){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;

    if(idx < Z_x_dim * Z_y_dim){
        d_Z2[idx] = 2 * (d_A2[idx] - d_one_hot_Y[idx]);
    }
}

void BackProp(float *d_Z1, float *d_A1, float *d_A2, float *d_W2, ParametersLinear *d_params1, ParametersLinear *d_params2, unsigned char *d_one_hot_Y, 
    float *d_data, float* d_dZ2, float *d_dZ1, int *h_numImages, int *h_numRows, int *h_numCols){
    
    printf("Start Back \n");
    hipError_t hipError_t;

    int batchSize = 1000;

    int numBatches = (*h_numImages) / batchSize;

    int batcharraysize = batchSize * 784;

    dim3 block_size(8,8);
    dim3 num_of_blocks((batchSize+block_size.x-1)/block_size.x,(*h_numRows * *h_numCols+block_size.y-1)/block_size.y);

    int singleDimblockSize = 1028;
    int singleDimnumBlocks = (batcharraysize + singleDimblockSize - 1) / singleDimblockSize;

    for (int i = 0; i < numBatches; i++) {
        printf("Batch %i Start \n", i);
        // Calculate the start and end indices for this batch
        int startIdx = i * batchSize;
        int endIdx = min(startIdx + batchSize, *h_numImages);

        // Calculate the size of this batch
        int batchMatrixSize = (endIdx - startIdx) * 784;

        startBackProp<<<singleDimblockSize, singleDimnumBlocks>>>(d_dZ2 + startIdx * 10, d_A2 + startIdx * 10, d_one_hot_Y, 60000, 10);
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (startBackProp): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }

        linearBackProp<<<num_of_blocks, block_size>>>(d_dZ2 + startIdx * 784, d_dZ1 + startIdx * 10, d_params2, 60000, 10, 10);
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearBackProp): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        
        reLUBack<<<singleDimblockSize, singleDimnumBlocks>>>(d_Z1 + startIdx * 784, d_dZ1 + startIdx * 10, 60000, 10);
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (reLUBack): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        // Linear 2
        linearUpdateWeight<<<num_of_blocks, block_size>>>(d_A1 + startIdx * 784, d_dZ2 + startIdx * 10, d_params2, 10, 10, 60000, 10, 60000);
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearUpdateWeight2): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        linearUpdateBias<<<singleDimblockSize, singleDimnumBlocks>>>(d_dZ2 + startIdx * 784, d_params2, 60000, 10);
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearUpdateWeight2): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        // Linear 1        
        linearUpdateWeight<<<num_of_blocks, block_size>>>(d_data + startIdx * 784, d_dZ1 + startIdx * 10, d_params1, 784, 10, 784, 10, 60000);
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearUpdateWeight2): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        linearUpdateBias<<<singleDimblockSize, singleDimnumBlocks>>>(d_dZ1 + startIdx * 784, d_params2, 60000, 10);
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearUpdateWeight2): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
    }
}   

__global__ void one_hot_encode(unsigned char* labels, unsigned char* output, int numLabels, int numClasses) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < numLabels) {
        // Initialize the output array for this label to zeros
        for (int i = 0; i < numClasses; i++) {
            output[idx * numClasses + i] = 0;
        }

        // Set the element at the index corresponding to the label to 1
        int label = labels[idx];
        if (label < numClasses) {
            output[idx * numClasses + label] = 1;
        }
    }
}

void NeuralNetwork(float *h_data, int *h_numImages, int *h_numRows, int *h_numCols, unsigned char *h_labels){

    printf("Init Params \n");
    ParametersLinear* h_params1 = (ParametersLinear*)malloc(sizeof(ParametersLinear));
    ParametersLinear* h_params2 = (ParametersLinear*)malloc(sizeof(ParametersLinear));


    float *d_data;
    int *d_numImages, *d_numRows, *d_numCols;
    ParametersLinear *d_params1, *d_params2;

    hipMalloc((void**)&d_data, *h_numImages* *h_numRows * *h_numCols * sizeof(float));

    hipMalloc((void**)&d_numImages, sizeof(int));
    hipMalloc((void**)&d_numRows, sizeof(int));
    hipMalloc((void**)&d_numCols, sizeof(int));

    /// Parameters Memory
    hipMalloc((void**)&d_params1, sizeof(ParametersLinear));
    hipMalloc((void**)&d_params2, sizeof(ParametersLinear));
  
    float* d_W1;
    hipMalloc((void**)&d_W1, 10 * 784 * sizeof(float));
    hipMemcpy(&(d_params1->W), &d_W1, sizeof(float*), hipMemcpyHostToDevice);

    float* d_B1;
    hipMalloc((void**)&d_B1, 10 * 1 * sizeof(float));
    hipMemcpy(&(d_params1->B), &d_B1, sizeof(float*), hipMemcpyHostToDevice);
    
    
    float* d_W2;
    hipMalloc((void**)&d_W2, 10 * 10 * sizeof(float));
    hipMemcpy(&(d_params2->W), &d_W2, sizeof(float*), hipMemcpyHostToDevice);
    float* d_B2;
    hipMalloc((void**)&d_B2, 10 * 1 * sizeof(float));
    hipMemcpy(&(d_params2->B), &d_B2, sizeof(float*), hipMemcpyHostToDevice);
    
    ///
    
    hipMemcpy(d_data, h_data, *h_numImages* *h_numRows * *h_numCols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_numImages, h_numImages, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numRows, h_numRows, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numCols, h_numCols, sizeof(int), hipMemcpyHostToDevice);
    
    
    /// Init the outputs of the forward steps
    float *d_Z1; // Z1
    hipMalloc((void**)&d_Z1, *h_numImages * 10 * sizeof(float));
    float *d_A1; // A1
    hipMalloc((void**)&d_A1, *h_numImages * 10 * sizeof(float));
    float *d_Z2; // Z2
    hipMalloc((void**)&d_Z2, *h_numImages * 10 * sizeof(float));
    float *d_A2; // A2
    hipMalloc((void**)&d_A2, *h_numImages * 10 * sizeof(float));

    float *d_dZ1; // Z1
    hipMalloc((void**)&d_Z1, *h_numImages * 10 * sizeof(float));
    float *d_dZ2; // Z2
    hipMalloc((void**)&d_Z2, *h_numImages * 10 * sizeof(float));

    unsigned char *d_labels;
    unsigned char *d_one_hot;

    int numLabels = 60000;
    int numClasses = 10; 

    hipMalloc((void**)&d_labels, numLabels * sizeof(unsigned char*));
    hipMalloc((void**)&d_one_hot, numLabels * numClasses * sizeof(unsigned char*));

    hipMalloc((void**)&d_labels, numLabels* sizeof(unsigned char*));

    int numThreads = 512;
    int numBlocks = (numLabels + numThreads - 1) / numThreads;
    hipError_t hipError_t;
    one_hot_encode<<<numBlocks, numThreads>>>(d_labels, d_one_hot, numLabels, numClasses);
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (one_hot_encode): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }

    // Testing with one forward prop
    ForwardProp(d_data, d_params1, d_params2, d_numImages, d_numRows, d_numCols, h_numImages, h_numRows, h_numCols,
                d_Z1, d_A1, d_Z2, d_A2);


    BackProp(d_Z1, d_A1, d_A2, d_W2, d_params1, d_params2, d_one_hot, d_data, d_dZ2, d_dZ1, h_numImages, h_numRows, h_numCols);
}