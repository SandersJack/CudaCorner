#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

extern "C" {
    #include "NeuralNetwork.h"
}

__global__ void linearForwardProp(float* A, float* Z, ParametersLinear *params, int *num_images, int *num_rows, int *num_cols){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int A_x_dim = 60000;
    int A_y_dim = 784;

    int W_x_dim = 10;
    int W_y_dim = A_y_dim;

    int Z_x_dim = A_x_dim;
    int Z_y_dim = W_y_dim;

    float Z_value = 0;

    if(idx < Z_x_dim && idy < Z_y_dim){
        for(int t=0; t< W_x_dim; t++){
            Z_value += A[idx*A_y_dim + idy] * params->W[t * W_y_dim + idy] + params->B[t];
        }
        
        Z[idx * Z_y_dim + idy] = Z_value;
    }
}

__global__ void linearUpdateWeight(float* A, float* dZ, ParametersLinear *params){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int A_x_dim = 60000;
    int A_y_dim = 784;

    int W_x_dim = 10;
    int W_y_dim = A_y_dim;

    int dZ_x_dim = 10;
    int dZ_y_dim = A_y_dim;

    float dW_value = 0.0f;

    float learning_rate = 0.01;

    if(idx < W_x_dim && idy < W_y_dim){
        for(int i=0; i<dZ_x_dim; i++){
            dW_value += dZ[idx*W_y_dim + i] * A[idy * A_y_dim + i];
        }

        params->W[idx * W_y_dim + idy] = params->W[idx * W_y_dim + idy] - learning_rate * dW_value/A_x_dim;
    }
}

__global__ void linearUpdateBias(float *dZ, ParametersLinear *params){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int dZ_x_dim = 10;
    int dZ_y_dim = 60000;

    float learning_rate = 0.01;

    if(idx < dZ_x_dim * dZ_y_dim){
        int dZ_x = idx / dZ_x_dim;
        int dZ_y = idx % dZ_x_dim;
        atomicAdd(&params->B[dZ_y], -learning_rate * (dZ[dZ_x * dZ_y_dim + dZ_y] / dZ_y_dim));
    }
}

void ForwardProp(float *d_A, ParametersLinear *d_params, int *d_numImages, int *d_numRows, int *d_numCols, int *h_numImages, int *h_numRows, int *h_numCols){
    float *d_Z;
    // Z (A.x, W.y)
    hipMalloc((void**)&d_Z, *h_numImages * *h_numRows * *h_numCols * sizeof(float));

    hipError_t hipError_t;

    dim3 block_size(8,8);
    dim3 num_of_blocks((*h_numImages+block_size.x-1)/block_size.x,(*h_numRows * *h_numCols+block_size.y-1)/block_size.y);

    linearForwardProp<<<num_of_blocks, block_size>>>(d_A, d_Z, d_params, d_numImages, d_numRows, d_numCols);
    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (linearForwardProp): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }
    
}

void BackProp(){}

void NeuralNetwork(float *h_data, int *h_numImages, int *h_numRows, int *h_numCols){

    ParametersLinear* h_params = (ParametersLinear*)malloc(sizeof(ParametersLinear));

    float *d_data;
    int *d_numImages, *d_numRows, *d_numCols;
    ParametersLinear *d_params;

    hipMalloc((void**)&d_data, *h_numImages* *h_numRows * *h_numCols * sizeof(float));

    hipMalloc((void**)&d_numImages, sizeof(int));
    hipMalloc((void**)&d_numRows, sizeof(int));
    hipMalloc((void**)&d_numCols, sizeof(int));

    hipMalloc((void**)&d_params, sizeof(ParametersLinear));

    hipMemcpy(d_data, h_data, *h_numImages* *h_numRows * *h_numCols * sizeof(float), hipMemcpyHostToDevice);
    
    hipMemcpy(d_numImages, h_numImages, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numRows, h_numRows, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numCols, h_numCols, sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_params, h_params, sizeof(ParametersLinear), hipMemcpyHostToDevice);

    // Testing with one forward prop
    ForwardProp(d_data, d_params, d_numImages, d_numRows, d_numCols, h_numImages, h_numRows, h_numCols);
}