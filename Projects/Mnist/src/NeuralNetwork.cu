#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <math.h>

extern "C" {
    #include "NeuralNetwork.h"
}

__global__ void sumExp(float *Z2, float *sum_exp){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int Z_x_dim = 60000;
    int Z_y_dim = 10;

    if(idx < Z_x_dim*Z_y_dim){
        atomicAdd(sum_exp, exp(Z2[idx]));
        //printf("VAL: %i %f \n",idx,  Z2[idx]);
    }
}

__global__ void softMax(float *Z2, float *A2, float *sum_exp, 
    int Z_x_dim, int Z_y_dim){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < Z_x_dim*Z_y_dim){
        A2[idx] = exp(Z2[idx]) * *sum_exp;
    }
}

__global__ void reLUBack(float* Z1, float* dA1, int Z_x_dim, int Z_y_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < Z_x_dim * Z_y_dim) {
        if(Z1[idx] > 0){
            dA1[idx] *= Z1[idx];
        } else {
            dA1[idx] *= 0;
        }
    }
}

__global__ void reLUForward(float *Z1, float *A1, int Z_x_dim, int Z_y_dim){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < Z_x_dim * Z_y_dim){
        if(Z1[idx] > 0){
            //A1[idx] = Z1[idx];
        } else {
            A1[0] = 0;
        }
    }
}

__global__ void linearForwardProp(float* A, float* Z, ParametersLinear *params, int *num_images, int *num_rows, int *num_cols,
    int Z_x_dim, int Z_y_dim, int W_x_dim, int W_y_dim){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float Z_value = 0;
    
    if(idx < Z_x_dim && idy < Z_y_dim){
        for(int t=0; t< W_y_dim; t++){
            Z_value += A[idx*W_y_dim + t] * params->W[idy * W_y_dim + t] + params->B[idy];
        }
        Z[idx * W_x_dim + idy] = Z_value;
    }
}

__global__ void linearBackProp(float *dZ2, float *dZ1, ParametersLinear *params, int Z_x_dim, int Z_y_dim, int W_y_dim){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float dZ1_value = 0.0f;

    if(idx < Z_x_dim && idy < Z_y_dim){
        for(int i=0; i<W_y_dim; i++){
            dZ1_value += params->W[i*W_y_dim + idx] * dZ2[i*W_y_dim + idy];
        }
        dZ2[idx*Z_y_dim+idy] = dZ1_value;
    }
}

__global__ void linearUpdateWeight(float* A, float* dZ, ParametersLinear *params, int W_x_dim, int W_y_dim, int A_x_dim, int A_y_dim, int dZ_x_dim){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float dW_value = 0.0f;

    float learning_rate = 0.01;

    if(idx < W_x_dim && idy < W_y_dim){
        for(int i=0; i<dZ_x_dim; i++){
            dW_value += dZ[idx*W_y_dim + i] * A[idy * A_y_dim + i];
        }

        params->W[idx * W_y_dim + idy] = params->W[idx * W_y_dim + idy] - learning_rate * dW_value/A_x_dim;
    }
}

__global__ void linearUpdateBias(float *dZ, ParametersLinear *params, int dZ_x_dim, int dZ_y_dim){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float learning_rate = 0.01;

    if(idx < dZ_x_dim * dZ_y_dim){
        int dZ_x = idx / dZ_x_dim;
        int dZ_y = idx % dZ_x_dim;
        atomicAdd(&params->B[dZ_y], -learning_rate * (dZ[dZ_x * dZ_y_dim + dZ_y] / dZ_y_dim));
    }
}

void ForwardProp(float *d_X, ParametersLinear *d_params1, ParametersLinear *d_params2, int *d_numImages, int *d_numRows, 
    int *d_numCols, int *h_numImages, int *h_numRows, int *h_numCols,
    float *d_Z1, float *d_A1, float *d_Z2, float *d_A2){

    float *d_sum_exp; 
    hipMalloc((void**)&d_sum_exp, sizeof(float));

    int matrixSize = *h_numImages * 784;
    printf("Start Forward \n");
    hipError_t hipError_t;

    int batchSize = 1000;

    int numBatches = (*h_numImages) / batchSize;

    int batcharraysize = batchSize * 784;

    dim3 block_size(16,16);
    dim3 num_of_blocks((batchSize+block_size.x-1)/block_size.x, (*h_numRows * *h_numCols+block_size.y-1)/block_size.y);

    dim3 num_of_blocks2((batchSize+block_size.x-1)/block_size.x, (10+block_size.y-1)/block_size.y);

    int singleDimblockSize = 1028;
    int singleDimnumBlocks = (batcharraysize + singleDimblockSize - 1) / singleDimblockSize;

    int singleDimnumBlocks2 = (batchSize * 10 + singleDimblockSize - 1) / singleDimblockSize;

    printf("Matrix Size %i \n", matrixSize);
    printf("Num block x threads %i \n", singleDimblockSize * singleDimnumBlocks);
    printf("Num Threads %i \n", singleDimnumBlocks);

    for (int i = 0; i < numBatches; i++) {
        printf("Batch %i Start \n", i);
        // Calculate the start and end indices for this batch
        int startIdx = i * batchSize;
        int endIdx = min(startIdx + batchSize, *h_numImages);
        // 47040000
        // 10976000
        // Calculate the size of this batch
        int batchMatrixSize = (endIdx - startIdx) * 784;
        printf("%i \n", startIdx * 784);
        /// First Linear Layer
        linearForwardProp<<<num_of_blocks, block_size>>>(d_X, d_Z1, d_params1, d_numImages, d_numRows, d_numCols,
            batchSize, 10, 10 , 784);
        hipDeviceSynchronize();
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearForwardProp1): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        /// Relu Layer
        reLUForward<<<singleDimblockSize, singleDimnumBlocks>>>(d_Z1, d_A1, batchSize, 10);
        hipDeviceSynchronize();

        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (reLUForward): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        /// Second Linear Layer
        linearForwardProp<<<num_of_blocks2, block_size>>>(d_A1, d_Z2, d_params2, d_numImages, d_numRows, d_numCols, 
            batchSize, 10, 10, 10);
        hipDeviceSynchronize();
        
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearForwardProp2): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        
        /// Sum the exponetial 
        sumExp<<<singleDimblockSize, singleDimnumBlocks2>>>(d_Z2, d_sum_exp);
        hipDeviceSynchronize();
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (sumExp): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        /// Conduct the SoftMax
        softMax<<<singleDimblockSize, singleDimnumBlocks2>>>(d_Z2, d_A2, d_sum_exp, 
        1000, 10);
        hipDeviceSynchronize();

        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (softMax): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }

        /// Pointer arith
        d_X += batchSize * 784;
        d_Z1 += batchSize * 10;
        d_A1 += batchSize * 10;
        d_Z2 += batchSize * 10;
        d_A2 += batchSize * 10;
    }
}

__global__ void startBackProp(float *d_Z2, float *d_A2, unsigned char *d_one_hot_Y, int Z_x_dim, int Z_y_dim){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < Z_x_dim * Z_y_dim){
        d_Z2[idx] = 2 * (d_A2[idx] - d_one_hot_Y[idx]);
    }
}

void BackProp(float *d_Z1, float *d_A1, float *d_A2, float *d_W2, ParametersLinear *d_params1, ParametersLinear *d_params2, unsigned char *d_one_hot_Y, 
    float *d_data, float* d_dZ2, float *d_dZ1, int *h_numImages, int *h_numRows, int *h_numCols){
    
    printf("Start Back \n");
    hipError_t hipError_t;

    int batchSize = 1000;

    int numBatches = (*h_numImages) / batchSize;

    int batcharraysize = batchSize * 784;
    int batcharraysize10 = batchSize * 10;

    dim3 block_size(8,8);
    dim3 num_of_blocks((batchSize+block_size.x-1)/block_size.x,(*h_numRows * *h_numCols+block_size.y-1)/block_size.y);

    int singleDimblockSize = 1028;
    int singleDimnumBlocks784 = (batcharraysize + singleDimblockSize - 1) / singleDimblockSize;
    int singleDimnumBlocks10 = (batcharraysize10 + singleDimblockSize - 1) / singleDimblockSize;

    for (int i = 0; i < numBatches; i++) {
        printf("Batch %i Start \n", i);
        // Calculate the start and end indices for this batch
        int startIdx = i * batchSize;
        int endIdx = min(startIdx + batchSize, *h_numImages);

        // Calculate the size of this batch
        int batchMatrixSize = (endIdx - startIdx) * 784;
        printf("%i \n", startIdx * 10);

        startBackProp<<<singleDimblockSize, singleDimnumBlocks10>>>(d_dZ2, d_A2, d_one_hot_Y, 60000, 10);
        hipDeviceSynchronize();
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (startBackProp): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }

        linearBackProp<<<num_of_blocks, block_size>>>(d_dZ2, d_dZ1, d_params2, 60000, 10, 10);
        hipDeviceSynchronize();
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearBackProp): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        
        reLUBack<<<singleDimblockSize, singleDimnumBlocks10>>>(d_Z1, d_dZ1, 60000, 10);
        hipDeviceSynchronize();
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (reLUBack): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        // Linear 2
        linearUpdateWeight<<<num_of_blocks, singleDimnumBlocks10>>>(d_A1, d_dZ2, d_params2, 10, 10, 60000, 10, 60000);
        hipDeviceSynchronize();
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearUpdateWeight2): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        linearUpdateBias<<<singleDimblockSize, singleDimnumBlocks10>>>(d_dZ2, d_params2, 60000, 10);
        hipDeviceSynchronize();
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearUpdateWeight2): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        // Linear 1        
        linearUpdateWeight<<<num_of_blocks, block_size>>>(d_data, d_dZ1, d_params1, 784, 10, 784, 10, 60000);
        hipDeviceSynchronize();
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearUpdateWeight2): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }
        linearUpdateBias<<<singleDimblockSize, singleDimnumBlocks10>>>(d_dZ1, d_params2, 60000, 10);
        hipDeviceSynchronize();
        hipError_t = hipGetLastError();
        if (hipError_t != hipSuccess) {
            printf("Kernel launch error (linearUpdateWeight2): %s\n", hipGetErrorString(hipError_t));
            // Handle error appropriately
            exit(0);
        }

        /// Pointer arith
        d_dZ2 += batchSize * 10;
        d_A2 += batchSize * 10;
        d_one_hot_Y += batchSize * 10;
        d_dZ1 += batchSize * 10;
        d_Z1 += batchSize * 10;
        d_A1 += batchSize * 10;
        d_data += batchSize * 784;
    }
}   

__global__ void one_hot_encode(unsigned char* labels, unsigned char* output, int numLabels, int numClasses) {
    int idx = threadIdx.x * blockIdx.x * blockDim.x;

    if (idx < numLabels) {
        // Initialize the output array for this label to zeros
        for (int i = 0; i < numClasses; i++) {
            output[idx * numClasses + i] = 0;
        }

        // Set the element at the index corresponding to the label to 1
        int label = labels[idx];
        if (label < numClasses) {
            output[idx * numClasses + label] = 1;
        }
    }
}

__device__ float getRandomNumber(hiprandState_t globalState) {
    float random = hiprand_uniform(&globalState) - 0.5f;
    return random;
}

__global__ void initParams(ParametersLinear *params1, ParametersLinear *params2){
    

    for(int i=0; i<10; i++){
        for(int j=0; j<784; j++){
            hiprandState_t globalState;
            hiprand_init(clock64() * i*j, 0, 0, &globalState);
            params1->W[i*784 + j] = getRandomNumber(globalState) / sqrtf(1./784.);
            //printf("VAL: %i %f \n", i*784 + j, params1->W[i*784 + j]);
        }
    }
    for(int i=0; i<10; i++){
        hiprandState_t globalState;
        hiprand_init(clock64() + i, 0, 0, &globalState);
        params1->B[i] = getRandomNumber(globalState) / sqrtf(1./10.);
    }
    for(int i=0; i<10; i++){
        for(int j=0; j<10; j++){
            hiprandState_t globalState;
            hiprand_init(clock64(),0 , 0, &globalState);
            params2->W[i*10 + j] = getRandomNumber(globalState) / sqrtf(1./20.);
        }
    }
    for(int i=0; i<10; i++){
        hiprandState_t globalState;
        hiprand_init(clock64()*i, 0, 0, &globalState);
        params2->B[i] = getRandomNumber(globalState) / sqrtf(1./10);
    }
}

__global__ void getPrediction(float *A2, int Z_x_dim, int Z_y_dim, int *predictions) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < Z_x_dim) {
        float max_val = A2[idx];
        int max_idx = idx;
        for (int i = idx * Z_y_dim; i < Z_x_dim * Z_y_dim; i++) {
            if (A2[i] > max_val) {
                max_val = A2[i];
                max_idx = idx;
            }
        }
        predictions[max_idx / 10] = max_idx % Z_y_dim;
    }
}

__global__ void _getAccuracy(int *predictions, unsigned char *Y, int numLabels, float *accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numLabels) {
        *accuracy += (predictions[idx] == Y[idx]) ? 1.0f : 0.0f;
    }
}

void getAccuracy(float *d_A2, unsigned char *d_labels, int *d_numImages){
    float *d_accuracy;
    int *d_predictions;
    hipError_t err;
    printf("Start Acc \n");
    hipMalloc((void **)&d_accuracy, sizeof(float));
    hipMalloc((void **)&d_predictions, 60000 * sizeof(int));

    printf("Start 1 \n");
    int numThreads = 512;
    int numBlocks = (10 + numThreads - 1) / numThreads;
    hipError_t hipError_t;
    printf("Start 2 \n");
    getPrediction<<<numBlocks, numThreads>>>(d_A2, *d_numImages, 10, d_predictions);
    hipDeviceSynchronize();
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (getPrediction): %s\n", hipGetErrorString(hipError_t));
        exit(0);
    }
    printf("Start 3 \n");
    _getAccuracy<<<numBlocks, numThreads>>>(d_predictions, d_labels, *d_numImages, d_accuracy);
    hipDeviceSynchronize();
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (_getAccuracy): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }

    float *h_accuracy = (float*)malloc(sizeof(float));
    hipMemcpy(h_accuracy, d_accuracy, sizeof(float), hipMemcpyDeviceToHost);

    printf("Accuracy: %f \n", *h_accuracy / 60000);
}

void NeuralNetwork(float *h_data, int *h_numImages, int *h_numRows, int *h_numCols, unsigned char *h_labels){

    printf("Init Params \n");
    ParametersLinear* h_params1 = (ParametersLinear*)malloc(sizeof(ParametersLinear));
    ParametersLinear* h_params2 = (ParametersLinear*)malloc(sizeof(ParametersLinear));


    float *d_data;
    int *d_numImages, *d_numRows, *d_numCols;
    ParametersLinear *d_params1, *d_params2;
    hipMalloc((void**)&d_data, *h_numImages* *h_numRows * *h_numCols * sizeof(float));

    hipMalloc((void**)&d_numImages, sizeof(int));
    hipMalloc((void**)&d_numRows, sizeof(int));
    hipMalloc((void**)&d_numCols, sizeof(int));

    /// Parameters Memory
    hipMalloc((void**)&d_params1, sizeof(ParametersLinear));
    hipMalloc((void**)&d_params2, sizeof(ParametersLinear));
  
    float* d_W1;
    hipMalloc((void**)&d_W1, 10 * 784 * sizeof(float));
    hipMemcpy(&(d_params1->W), &d_W1, sizeof(float*), hipMemcpyHostToDevice);

    float* d_B1;
    hipMalloc((void**)&d_B1, 10 * 1 * sizeof(float));
    hipMemcpy(&(d_params1->B), &d_B1, sizeof(float*), hipMemcpyHostToDevice);
    
    
    float* d_W2;
    hipMalloc((void**)&d_W2, 10 * 10 * sizeof(float));
    hipMemcpy(&(d_params2->W), &d_W2, sizeof(float*), hipMemcpyHostToDevice);
    float* d_B2;
    hipMalloc((void**)&d_B2, 10 * 1 * sizeof(float));
    hipMemcpy(&(d_params2->B), &d_B2, sizeof(float*), hipMemcpyHostToDevice);

    hipError_t hipError_t;
    /// Init the device weights

    initParams<<<1, 1>>>(d_params1, d_params2);
    hipDeviceSynchronize();
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (initParams): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }
    
    ///
    
    hipMemcpy(d_data, h_data, *h_numImages* *h_numRows * *h_numCols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_numImages, h_numImages, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numRows, h_numRows, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numCols, h_numCols, sizeof(int), hipMemcpyHostToDevice);
    
    
    /// Init the outputs of the forward steps
    float *d_Z1; // Z1
    hipMalloc((void**)&d_Z1, *h_numImages * 10 * sizeof(float));
    float *d_A1; // A1
    hipMalloc((void**)&d_A1, *h_numImages * 10 * sizeof(float));
    float *d_Z2; // Z2
    hipMalloc((void**)&d_Z2, *h_numImages * 10 * sizeof(float));
    float *d_A2; // A2
    hipMalloc((void**)&d_A2, *h_numImages * 10 * sizeof(float));

    float *d_dZ1; // Z1
    hipMalloc((void**)&d_dZ1, *h_numImages * 10 * sizeof(float));
    float *d_dZ2; // Z2
    hipMalloc((void**)&d_dZ2, *h_numImages * 10 * sizeof(float));

    unsigned char *d_labels;
    unsigned char *d_one_hot;

    int numLabels = 60000;
    int numClasses = 10; 


    hipMalloc((void**)&d_one_hot, numLabels * numClasses * sizeof(unsigned char));

    hipMalloc((void**)&d_labels, numLabels * sizeof(unsigned char));
    hipMemcpy(d_labels, h_labels, numLabels * sizeof(unsigned char), hipMemcpyHostToDevice);


    int numThreads = 512;
    int numBlocks = (numLabels + numThreads - 1) / numThreads;
    
    one_hot_encode<<<numBlocks, numThreads>>>(d_labels, d_one_hot, numLabels, numClasses);
    hipDeviceSynchronize();
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (one_hot_encode): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }
    float *d_A1_orgininal = d_A1;
    float *d_A2_orgininal = d_A2;
    float *d_Z1_orgininal = d_Z1;
    float *d_Z2_orgininal = d_Z2;

    float *d_data_original = d_data;
    // Testing with one forward prop
    ForwardProp(d_data, d_params1, d_params2, d_numImages, d_numRows, d_numCols, h_numImages, h_numRows, h_numCols,
                d_Z1, d_A1, d_Z2, d_A2);

    d_A1 = d_A1_orgininal; 
    d_A2 = d_A2_orgininal; 
    d_Z1 = d_Z1_orgininal; 
    d_Z2 = d_Z2_orgininal;

    d_data = d_data_original;
    
    BackProp(d_Z1, d_A1, d_A2, d_W2, d_params1, d_params2, d_one_hot, d_data, d_dZ2, d_dZ1, h_numImages, h_numRows, h_numCols);

    d_A1 = d_A1_orgininal; 
    d_A2 = d_A2_orgininal; 
    d_Z1 = d_Z1_orgininal; 
    d_Z2 = d_Z2_orgininal;

    d_data = d_data_original;
    getAccuracy(d_A2, d_labels, d_numImages);
}