#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <math.h>

extern "C" {
    #include "NeuralNetwork.h"
}

__global__ void sumExp(float *Z2, float *sum_exp){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int A_x_dim = 60000;
    int A_y_dim = 784;

    int Z_x_dim = 10;
    int Z_y_dim = A_x_dim;

    if(idx < Z_x_dim*Z_y_dim){
        atomicAdd(sum_exp, exp(Z2[idx]));
    }
}

__global__ void softMax(float *Z2, float *A2, float *sum_exp){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int A_x_dim = 60000;
    int A_y_dim = 784;

    int Z_x_dim = 10;
    int Z_y_dim = A_x_dim;

    if(idx < Z_x_dim*Z_y_dim){
        A2[idx] = exp(Z2[idx]) / *sum_exp;
    }
}

__global__ void reLUBack(float* Z1, float* dA1, float* dZ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int A_x_dim = 60000;
    int A_y_dim = 784;

    int W_x_dim = 10;
    int W_y_dim = A_y_dim;

    int Z_x_dim = W_x_dim;
    int Z_y_dim = A_x_dim;

    if(idx < Z_x_dim * Z_y_dim) {
        if(Z1[idx] > 0){
            dA1[idx] *= Z1[idx];
        } else {
            dA1[idx] *= 0;
        }
    }
}

__global__ void reLUForward(float *Z1, float *A1){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;

    int A_x_dim = 60000;
    int A_y_dim = 784;

    int W_x_dim = 10;
    int W_y_dim = A_y_dim;

    int Z_x_dim = W_x_dim;
    int Z_y_dim = A_x_dim;

    if(idx < Z_x_dim * Z_y_dim){
        if(Z1[idx] > 0){
            A1[idx] = Z1[idx];
        } else {
            A1[idx] = 0;
        }
    }
}

__global__ void linearForwardProp(float* A, float* Z, ParametersLinear *params, int *num_images, int *num_rows, int *num_cols){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int A_x_dim = 60000;
    int A_y_dim = 784;

    int W_x_dim = 10;
    int W_y_dim = A_y_dim;

    int Z_x_dim = A_x_dim;
    int Z_y_dim = W_x_dim;

    float Z_value = 0;

    float *W = params->W;
    float *B = params->B; 

    //float val = params->W[0];

    if(idx < Z_x_dim && idy < Z_y_dim){
        for(int t=0; t< W_y_dim; t++){
            Z_value += A[idx*A_y_dim + t] * params->W[t * W_y_dim + idx] + params->B[idy];
        }
        
        Z[idx * Z_y_dim + idy] = Z_value;
    }
}

__global__ void linearBackProp(float *dZ2, ParametersLinear *params, float *dZ1){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int dA_x_dim = 10;
    int dA_y_dim = 60000;

    int A_x_dim = 60000;
    int A_y_dim = 784;

    int W_x_dim = 10;
    int W_y_dim = A_y_dim;

    float dZ1_value = 0.0f;

    if(idx < dA_x_dim && idy < dA_y_dim){
        for(int i=0; i<W_y_dim; i++){
            dZ1_value += params->W[i*W_y_dim + idx] * dZ2[i*W_y_dim + idy];
        }
        dZ2[idx*dA_y_dim+idy] = dZ1_value;
    }
}

__global__ void linearUpdateWeight(float* A, float* dZ, ParametersLinear *params){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int A_x_dim = 60000;
    int A_y_dim = 784;

    int W_x_dim = 10;
    int W_y_dim = A_y_dim;

    int dZ_x_dim = 10;
    int dZ_y_dim = A_y_dim;

    float dW_value = 0.0f;

    float learning_rate = 0.01;

    if(idx < W_x_dim && idy < W_y_dim){
        for(int i=0; i<dZ_x_dim; i++){
            dW_value += dZ[idx*W_y_dim + i] * A[idy * A_y_dim + i];
        }

        params->W[idx * W_y_dim + idy] = params->W[idx * W_y_dim + idy] - learning_rate * dW_value/A_x_dim;
    }
}

__global__ void linearUpdateBias(float *dZ, ParametersLinear *params){
    int idx = blockIdx.x + blockDim.x + threadIdx.x;
    
    int dZ_x_dim = 10;
    int dZ_y_dim = 60000;

    float learning_rate = 0.01;

    if(idx < dZ_x_dim * dZ_y_dim){
        int dZ_x = idx / dZ_x_dim;
        int dZ_y = idx % dZ_x_dim;
        atomicAdd(&params->B[dZ_y], -learning_rate * (dZ[dZ_x * dZ_y_dim + dZ_y] / dZ_y_dim));
    }
}

void ForwardProp(float *d_X, ParametersLinear *d_params1, ParametersLinear *d_params2, int *d_numImages, int *d_numRows, 
    int *d_numCols, int *h_numImages, int *h_numRows, int *h_numCols,
    float *d_Z1, float *d_A1, float *d_Z2, float *d_A2){

    int matrixSize = *h_numImages * 784;
    printf("Start Forward \n");
    hipError_t hipError_t;

    dim3 block_size(8,8);
    dim3 num_of_blocks((*h_numImages+block_size.x-1)/block_size.x,(*h_numRows * *h_numCols+block_size.y-1)/block_size.y);

    int singleDimblockSize = 1024;
    int singleDimnumBlocks = (matrixSize + singleDimblockSize - 1) / singleDimblockSize;
    printf("Matrix Size %i \n", matrixSize);
    printf("Num block x threads %i \n", singleDimblockSize * singleDimnumBlocks);
    printf("Num Threads %i \n", singleDimnumBlocks);
    /// First Linear Layer
    linearForwardProp<<<num_of_blocks, block_size>>>(d_X, d_Z1, d_params1, d_numImages, d_numRows, d_numCols);
    hipDeviceSynchronize();
    printf("Lin1 Done \n");
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (linearForwardProp1): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }
    /// Relu Layer
    reLUForward<<<singleDimblockSize, singleDimnumBlocks>>>(d_Z1, d_A1);
    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (reLUForward): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }
    /// Second Linear Layer
    linearForwardProp<<<num_of_blocks, block_size>>>(d_A1, d_Z2, d_params2, d_numImages, d_numRows, d_numCols);
    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (linearForwardProp2): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }

    float *d_sum_exp; 
    hipMalloc((void**)&d_sum_exp, sizeof(float));

    /// Sum the exponetial 
    sumExp<<<singleDimblockSize, singleDimnumBlocks>>>(d_Z2, d_sum_exp);
    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (softMax): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }
    /// Conduct the SoftMax
    softMax<<<singleDimblockSize, singleDimnumBlocks>>>(d_Z2, d_A2, d_sum_exp);
    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (softMax): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }

}

void BackProp(){}

void NeuralNetwork(float *h_data, int *h_numImages, int *h_numRows, int *h_numCols){

    printf("Init Params \n");
    ParametersLinear* h_params1 = (ParametersLinear*)malloc(sizeof(ParametersLinear));
    ParametersLinear* h_params2 = (ParametersLinear*)malloc(sizeof(ParametersLinear));

    //h_params1->W = (float*)malloc(10 * 784 * sizeof(float));
    //h_params1->B = (float*)malloc(10 * 1 * sizeof(float));


    float *d_data;
    int *d_numImages, *d_numRows, *d_numCols;
    ParametersLinear *d_params1, *d_params2;

    hipMalloc((void**)&d_data, *h_numImages* *h_numRows * *h_numCols * sizeof(float));

    hipMalloc((void**)&d_numImages, sizeof(int));
    hipMalloc((void**)&d_numRows, sizeof(int));
    hipMalloc((void**)&d_numCols, sizeof(int));

    /// Parameters Memory
    hipMalloc((void**)&d_params1, sizeof(ParametersLinear));
    //hipMalloc((void**)&d_params2, sizeof(ParametersLinear));
  
    float* d_W1;
    hipMalloc((void**)&d_W1, 10 * 784 * sizeof(float));
    hipMemcpy(&(d_params1->W), &d_W1, sizeof(float*), hipMemcpyHostToDevice);

    float* d_B1;
    hipMalloc((void**)&d_B1, 10 * 1 * sizeof(float));
    hipMemcpy(&(d_params1->B), &d_B1, sizeof(float*), hipMemcpyHostToDevice);
    
    /*
    float* d_W2;
    hipMalloc((void**)&d_W1, 10 * 10 * sizeof(float));
    hipMemcpy(&(d_params2->W), &d_W2, sizeof(float*), hipMemcpyHostToDevice);
    float* d_B2;
    hipMalloc((void**)&d_B2, 10 * 1 * sizeof(float));
    hipMemcpy(&(d_params2->B), &d_B2, sizeof(float*), hipMemcpyHostToDevice);
    */
    ///
    
    hipMemcpy(d_data, h_data, *h_numImages* *h_numRows * *h_numCols * sizeof(float), hipMemcpyHostToDevice);
    
    hipMemcpy(d_numImages, h_numImages, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numRows, h_numRows, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numCols, h_numCols, sizeof(int), hipMemcpyHostToDevice);
    
    
    /// Init the outputs of the forward steps
    float *d_Z1; // Z1 (A.x, W.y)
    hipMalloc((void**)&d_Z1, *h_numImages * 10 * sizeof(float));
    float *d_A1; // A1 (A.x, W.y)
    hipMalloc((void**)&d_A1, *h_numImages * 10 * sizeof(float));
    float *d_Z2; // Z2
    hipMalloc((void**)&d_Z2, *h_numImages * 10 * sizeof(float));
    float *d_A2; // A2
    hipMalloc((void**)&d_A2, *h_numImages * 10 * sizeof(float));

    // Testing with one forward prop
    ForwardProp(d_data, d_params1, d_params2, d_numImages, d_numRows, d_numCols, h_numImages, h_numRows, h_numCols,
                d_Z1, d_A1, d_Z2, d_A2);
}