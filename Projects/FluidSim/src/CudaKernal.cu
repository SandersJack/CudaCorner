#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

extern "C" {
    #include "CudaKernal.h"
}

int compare(const void *a, const void *b) {
    const SpacialIndex *pa = (const SpacialIndex *)a;
    const SpacialIndex *pb = (const SpacialIndex *)b;
    return (pa->key > pb->key) - (pa->key < pb->key);
}

__global__ void kernel() {
    printf("Hello from CUDA kernel!\n");
}

void cuda_kernel() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}

static __device__ float smoothingKernal_new(float radius, float dis){
    if( dis >= radius) return 0;

    float volume = (M_PI * pow(radius, 4)) / 6;
    return (radius - dis) * (radius - dis) / volume;
}

static __device__ float smoothingKernalDerivative_new(float radius, float dis){
    if(dis >= radius) return 0;

    float scale = 12 / (pow(radius,4) * M_PI);
    return (dis - radius) * scale;
}

static __device__ void applyGravity(Particle *particles, int id){
    particles[id].dy -= 0.1;
}

static __device__ void resolveCollisions(Particle *particles, int id){

    int L = 800 - 50;
    float DECAY = 0.5;

    if(particles[id].x > L){
        particles[id].x = L - 0.1;
        particles[id].dx *= -DECAY;
    } else if(particles[id].x < 50) {
        particles[id].x = 50 + 0.1;
        particles[id].dx *= -DECAY;
    }
    if(particles[id].y > L){
        particles[id].y = L - 0.1;
        particles[id].dy *= -DECAY;
    } else if(particles[id].y < 50){
        particles[id].y = 50 + 0.1;
        particles[id].dy *= -DECAY;
    }
}

__global__ void cuda_updateParticlePred(Particle *particles, float *dt, int *NUM_PARTICLES){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < *NUM_PARTICLES){

        particles[idx].pred_x = particles[idx].x + particles[idx].dx * *dt;
        particles[idx].pred_y = particles[idx].y - particles[idx].dy * *dt;

    }
}

__global__ void cuda_updateParticle(Particle *particles, float *densities, float *dt, int *NUM_PARTICLES, FloatPair *pressureForce){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < *NUM_PARTICLES){

        //printf("Densities %f \n", densities[idx]);
        float pressureAccel_X = pressureForce[idx].first ;// densities[idx];
        float pressureAccel_Y = pressureForce[idx].second ;// densities[idx];

        particles[idx].dx += pressureAccel_X * *dt;
        particles[idx].dy += pressureAccel_Y * *dt;

        //applyGravity(particles, idx);
        resolveCollisions(particles, idx);

        particles[idx].x += particles[idx].dx * *dt;
        particles[idx].y -= particles[idx].dy * *dt;

    }
}

static __device__ uIntPair posToCellCoord(Particle particle, float radius){
    uIntPair pair;
    pair.first = (uint)(particle.pred_x / radius);
    pair.second = (uint)(particle.pred_y / radius);
    return pair;
}

static __device__ uint hashCell(int cellX, int cellY){
    uint a = cellX * 15823;
    uint b = cellY * 9737333;
    return a + b;
}

static __device__ uint getKeyFromHash(int *NUM_PARTICLES, uint hash){
    uint val =  hash % *NUM_PARTICLES;
    return val;
}


void __global__ calculateDensities(float* densities, Particle *particles, int *NUM_PARTICLES, int* spatialLookup, SpacialIndex *spacialIndexs, IntPair *offsets, 
        float *smoothingRadius, float *MASS){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < *NUM_PARTICLES){
        uIntPair origin_cell = posToCellCoord(particles[idx], *smoothingRadius);
        float density = 0;
        //printf("%f \n", density);

        float sqrRadius = *smoothingRadius * *smoothingRadius;

        for(int i=0; i<9; i++){
            
            uint hash = hashCell(origin_cell.first + offsets[i].first, origin_cell.second + offsets[i].second);
		    uint key = getKeyFromHash(NUM_PARTICLES, hash);

		    uint currIndex = spatialLookup[key];

            //printf("Testing %i %i %i \n", hash, key, currIndex);

            while (currIndex < *NUM_PARTICLES)
		    {
                SpacialIndex indexData = spacialIndexs[currIndex];
                currIndex++;
                
                if(indexData.key != key) break;
                if(indexData.hash != hash) continue;

                int n_index = indexData.index;

                float dist_x = particles[n_index].pred_x - particles[idx].pred_x;
                float dist_y = particles[n_index].pred_y - particles[idx].pred_y;

                float sqrdist = (dist_x * dist_x + dist_y * dist_y);

                if(sqrdist >= sqrRadius) continue;
                float dist = sqrt(sqrdist);
                float influence = smoothingKernal_new(*smoothingRadius, dist);
                density += *MASS * influence;
                //printf("Desnities %f %f %f %f \n", *MASS,  influence, dist, sqrdist);
            }
        }
        densities[idx] = density;
        //printf("Desnities %f \n", densities[idx]);
    }
}
__global__ void updateSpacialLookup_step1(Particle *particles, int *spatialLookup, SpacialIndex *spacialIndexs, int *NUM_PARTICLES, float *smoothingRadius){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < *NUM_PARTICLES){
        uIntPair cell = posToCellCoord(particles[idx], *smoothingRadius);
        uint hash = hashCell(cell.first, cell.second);
        uint cellKey = getKeyFromHash(NUM_PARTICLES, hash);
        
        spatialLookup[idx] = *NUM_PARTICLES;

        spacialIndexs[idx].index = idx; spacialIndexs[idx].hash = hash; spacialIndexs[idx].key = cellKey; 
    }
}

__global__ void updateSpacialLookup_step2(Particle *particles, int *spatialLookup, SpacialIndex *spacialIndexs, int *NUM_PARTICLES, float *smoothingRadius){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    
    if(idx < *NUM_PARTICLES){
        int key = spacialIndexs[idx].key;
        int keyPrev = (idx == 0) ? *NUM_PARTICLES : spacialIndexs[idx-1].key;
        
        if(key != keyPrev){
            spatialLookup[key] = idx;

        }
    }
}


__device__ float convertDensityToPressure(float density, float PRESSURE_MULT, float TARGET_DENSITY){

    float densityError = density / TARGET_DENSITY;
    float diff = density - TARGET_DENSITY;
    if (diff < 0) densityError *= -1;

    float pressure =  densityError * PRESSURE_MULT;
    return pressure;
}


__device__ float GetRandomDir() {
    // Initialize hiprand state for the thread
    hiprandState localState;
    hiprand_init(clock64(), threadIdx.x, 0, &localState);

    // Generate a random floating-point value between -1 and 1
    return 2.0f * hiprand_uniform(&localState) - 1.0f;
}

__global__ void calculateDensityForces(float* densities, Particle *particles, int *NUM_PARTICLES, int* spatialLookup, 
    SpacialIndex *spacialIndexs, IntPair *offsets, FloatPair *pressureForces, float* smoothingRadius, float *MASS, 
    float *pressureMult, float *targetDensity){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if( idx < *NUM_PARTICLES){

        float pressure = convertDensityToPressure(densities[idx], *pressureMult, *targetDensity);

        FloatPair __pressureForce; __pressureForce.first = 0.; __pressureForce.second = 0.;
        uIntPair origin_cell = posToCellCoord(particles[idx], *smoothingRadius);


        float sqrRadius = *smoothingRadius * *smoothingRadius;

        for(int i=0; i<9; i++){
            uint hash = hashCell(origin_cell.first + offsets[i].first, origin_cell.second + offsets[i].second);
		    uint key = getKeyFromHash(NUM_PARTICLES, hash);
            
		    uint currIndex = spatialLookup[key];
            //printf("CurrIndex %i \n", currIndex);

            while (currIndex < *NUM_PARTICLES)
		    {
                //printf("Current Index %i \n", currIndex);
                
                SpacialIndex indexData = spacialIndexs[currIndex];
                currIndex++;
                

                if(indexData.key != key) break;
                if(indexData.hash != hash) continue;

                int n_index = indexData.index;

                
                //printf("Index %i \n", n_index);

                float dist_X = particles[n_index].pred_x - particles[idx].pred_x;
                float dist_Y = particles[n_index].pred_y - particles[idx].pred_y;
                float sqDist = (dist_X*dist_X + dist_Y*dist_Y);

                if(sqDist >= sqrRadius) continue;

                float dist = sqrt(sqDist);

                float dir_X = (dist <= 0) ? 0: dist_X / dist;
                float dir_Y = (dist <= 0) ? 1: dist_Y / dist;
                
                float n_density = densities[n_index];

                float neighbourPressure = convertDensityToPressure(n_density, *pressureMult, *targetDensity);
                //printf(" %i Density A and B %i %f  %i %f \n", currIndex, n_index, densities[n_index], idx, densities[idx]);
                float sharedPressure = (pressure + neighbourPressure) * 0.5;

                float influence = smoothingKernalDerivative_new(*smoothingRadius, dist);

                __pressureForce.first += dir_X * sharedPressure * influence / n_density;
                __pressureForce.second -= dir_Y * sharedPressure * influence / n_density;
                

            }
        }

        pressureForces[idx].first = __pressureForce.first; pressureForces[idx].second = __pressureForce.second; 
    }
}

void __updateParticle(Particle *h_particles, float *h_dt, int *h_NUM_PARTICLES, float *h_densities, int *h_spatialLookup, SpacialIndex *h_spacialIndexs, 
        FloatPair *h_pressureForce, IntPair *h_offsets, float *h_smoothingRadius, float *h_mass, float *h_pressureMult, float *h_targetDensity){
    // Allocate memory for the array on the GPU
    Particle *d_particles;
    int *d_NUM_PARTICLES, *d_spatialLookup;
    float *d_dt, *d_densities, *d_smoothingRadius, *d_mass, *d_pressureMult, *d_targetDensity;

    FloatPair *d_pressureForce;
    IntPair *d_offsets;
    SpacialIndex *d_spacialIndexs, *d_spacialIndexs2;

    hipError_t hipError_t;

    hipMalloc((void**)&d_particles, *h_NUM_PARTICLES * sizeof(Particle));
    hipMalloc((void**)&d_NUM_PARTICLES, sizeof(int));

    hipMalloc((void**)&d_spatialLookup, *h_NUM_PARTICLES * sizeof(int));

    hipMalloc((void**)&d_spacialIndexs, *h_NUM_PARTICLES * sizeof(SpacialIndex));
    hipMalloc((void**)&d_spacialIndexs2, *h_NUM_PARTICLES * sizeof(SpacialIndex));
    
    hipMalloc((void**)&d_targetDensity, sizeof(float));
    hipMalloc((void**)&d_pressureMult, sizeof(float));
    hipMalloc((void**)&d_mass, sizeof(float));
    hipMalloc((void**)&d_smoothingRadius, sizeof(float));
    hipMalloc((void**)&d_dt, sizeof(float));

    hipMalloc((void**)&d_densities, *h_NUM_PARTICLES * sizeof(float));
    hipMalloc((void**)&d_pressureForce, *h_NUM_PARTICLES * sizeof(FloatPair));
    hipMalloc((void**)&d_offsets, 9 * sizeof(IntPair));


    hipMemcpy(d_particles, h_particles, *h_NUM_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);
    hipMemcpy(d_NUM_PARTICLES, h_NUM_PARTICLES, sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_spatialLookup, h_spatialLookup, *h_NUM_PARTICLES * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_spacialIndexs, h_spacialIndexs, *h_NUM_PARTICLES * sizeof(SpacialIndex), hipMemcpyHostToDevice);
    
    hipMemcpy(d_targetDensity, h_targetDensity, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_pressureMult, h_pressureMult, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mass, h_mass, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_smoothingRadius, h_smoothingRadius, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dt, h_dt, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_densities, h_densities, *h_NUM_PARTICLES * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_pressureForce, h_pressureForce, *h_NUM_PARTICLES * sizeof(FloatPair), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, h_offsets, 9 * sizeof(IntPair), hipMemcpyHostToDevice);

    int blockSize = 512;
    int numBlocks = (*h_NUM_PARTICLES + blockSize - 1) / blockSize;

    cuda_updateParticlePred<<<numBlocks, blockSize>>>(d_particles, d_dt, d_NUM_PARTICLES);
    
    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (cuda_updateParticlePred): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }
    
    updateSpacialLookup_step1<<<numBlocks, blockSize>>>(d_particles, d_spatialLookup, d_spacialIndexs, d_NUM_PARTICLES, d_smoothingRadius);

    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (updateSpacialLookup_step1): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }
    
    hipMemcpy(h_spacialIndexs, d_spacialIndexs, *h_NUM_PARTICLES * sizeof(SpacialIndex), hipMemcpyDeviceToHost);


    hipDeviceSynchronize();

    int sizeOfSL = *h_NUM_PARTICLES; 
    qsort(h_spacialIndexs, sizeOfSL, sizeof(SpacialIndex), compare);

    //for(int i=0 ;i<*h_NUM_PARTICLES; i++){
    //    printf("%i %i %i \n", h_spacialIndexs[i].index, h_spacialIndexs[i].hash, h_spacialIndexs[i].key);
    //}

    hipMemcpy(d_spacialIndexs2, h_spacialIndexs, *h_NUM_PARTICLES * sizeof(SpacialIndex), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (copy): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }
    
    updateSpacialLookup_step2<<<numBlocks, blockSize>>>(d_particles, d_spatialLookup, d_spacialIndexs2, d_NUM_PARTICLES, d_smoothingRadius);
    
    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (updateSpacialLookup_step2): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }

    calculateDensities<<<numBlocks, blockSize>>>(d_densities, d_particles, d_NUM_PARTICLES, d_spatialLookup, d_spacialIndexs, d_offsets, d_smoothingRadius, d_mass);

    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (calculateDensities): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }

    calculateDensityForces<<<numBlocks, blockSize>>>(d_densities, d_particles, d_NUM_PARTICLES, d_spatialLookup, d_spacialIndexs, d_offsets, 
            d_pressureForce, d_smoothingRadius, d_mass, d_pressureMult, d_targetDensity);
    hipDeviceSynchronize();
    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Kernel launch error (calculateDensityForces): %s\n", hipGetErrorString(hipError_t));
        // Handle error appropriately
        exit(0);
    }
    
    cuda_updateParticle<<<numBlocks, blockSize>>>(d_particles, d_densities, d_dt, d_NUM_PARTICLES, d_pressureForce);

    hipDeviceSynchronize();

    hipMemcpy(h_particles, d_particles, *h_NUM_PARTICLES * sizeof(Particle), hipMemcpyDeviceToHost);
    hipMemcpy(h_densities, d_densities, *h_NUM_PARTICLES * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_spacialIndexs, d_spacialIndexs, *h_NUM_PARTICLES * sizeof(SpacialIndex), hipMemcpyDeviceToHost);
    hipMemcpy(h_spatialLookup, d_spatialLookup, *h_NUM_PARTICLES * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_pressureForce, d_pressureForce, *h_NUM_PARTICLES * sizeof(FloatPair), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    float avgDen = 0;
    for(int i=0; i<*h_NUM_PARTICLES; i++){
        avgDen += h_densities[i];
        //printf("Particles v %f %f \n", h_particles[i].dx, h_particles[i].dy);
        //printf("Densitie v %f \n", h_densities[i]);

    }
    
    printf("Average Density %f \n", avgDen / *h_NUM_PARTICLES);
    
    hipFree(d_particles);
    hipFree(d_NUM_PARTICLES);
    hipFree(d_spatialLookup);
    hipFree(d_spacialIndexs2);
    hipFree(d_dt);
    hipFree(d_densities);
    hipFree(d_pressureForce);
    hipFree(d_offsets);
    hipFree(d_spacialIndexs);
}