#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" {
    #include "CudaKernal.h"
}

__global__ void kernel() {
    printf("Hello from CUDA kernel!\n");
}

void cuda_kernel() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}

static __device__ float smoothingKernal_new(float radius, float dis){
    if( dis >= radius) return 0;
    
    float volume = (M_PI * pow(radius, 4)) / 6;
    return (radius - dis) * (radius - dis) / volume;
}

static __device__ float smoothingKernalDerivative_new(float radius, float dis){
    if(dis >= radius) return 0;

    float scale = 12 / (pow(radius,4) * M_PI);
    return (dis - radius) * scale;
}

static __device__ void applyGravity(Particle *particles, int id){
    particles[id].dy -= 1;
}

static __device__ void resolveCollisions(Particle *particles, int id){

    int L = 800 - 50;
    float DECAY = 0.5;

    if(particles[id].x > L){
        particles[id].x = L - 0.1;
        particles[id].dx *= -DECAY;
    } else if(particles[id].x < 50) {
        particles[id].x = 50 + 0.1;
        particles[id].dx *= -DECAY;
    }
    if(particles[id].y > L){
        particles[id].y = L - 0.1;
        particles[id].dy *= -DECAY;
    } else if(particles[id].y < 50){
        particles[id].y = 50 + 0.1;
        particles[id].dy *= -DECAY;
    }
}

__global__ void cuda_updateParticle(Particle *particles, float *dt, int *NUM_PARTICLES){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < *NUM_PARTICLES){

        particles[idx].pred_x = particles[idx].x + particles[idx].dx * *dt;
        particles[idx].pred_y = particles[idx].y - particles[idx].dy * *dt;

        applyGravity(particles, idx);
        resolveCollisions(particles, idx);

        particles[idx].x += particles[idx].dx * *dt;
        particles[idx].y -= particles[idx].dy * *dt;

    }
}

void __global__ calculateDensities(float* densities, Particle *particles, int *num_particles){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int MASS = 1;

    if(idx < *num_particles &&  idy < *num_particles){

        float dist_x = particles[idx].pred_x - particles[idy].pred_x;
        float dist_y = particles[idx].pred_y - particles[idy].pred_y;

        float dist = sqrt(dist_x * dist_x + dist_y * dist_y);
        float influence = smoothingKernal_new(100, dist);
        densities[idx] += MASS * influence;

    }
}

void __updateParticle(Particle *h_particles, float *h_dt, int *h_NUM_PARTICLES, float *h_densities){
    // Allocate memory for the array on the GPU
    Particle *d_particles;
    int *d_NUM_PARTICLES;
    float *d_dt, *d_densities;
    hipMalloc((void**)&d_particles, *h_NUM_PARTICLES * sizeof(Particle));
    hipMalloc((void**)&d_NUM_PARTICLES, sizeof(int));
    hipMalloc((void**)&d_dt, sizeof(float));
    hipMalloc((void**)&d_densities, *h_NUM_PARTICLES * sizeof(float));

    hipMemcpy(d_particles, h_particles, *h_NUM_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);
    hipMemcpy(d_NUM_PARTICLES, h_NUM_PARTICLES, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dt, h_dt, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_densities, h_densities, *h_NUM_PARTICLES * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (*h_NUM_PARTICLES + blockSize - 1) / blockSize;
    cuda_updateParticle<<<numBlocks, blockSize>>>(d_particles, d_dt, d_NUM_PARTICLES);

    dim3 blockSize2(512, 512); 
    dim3 gridSize2((*h_NUM_PARTICLES + blockSize2.x - 1) / blockSize2.x, (*h_NUM_PARTICLES + blockSize2.y - 1) / blockSize2.y);

    calculateDensities<<<gridSize2, blockSize2>>>(d_densities, d_particles, d_NUM_PARTICLES);
    
    hipDeviceSynchronize();

    hipMemcpy(h_particles, d_particles, *h_NUM_PARTICLES * sizeof(Particle), hipMemcpyDeviceToHost);
    hipMemcpy(h_densities, d_densities, *h_NUM_PARTICLES * sizeof(float), hipMemcpyDeviceToHost);


    hipFree(d_particles);
    hipFree(d_densities);
    hipFree(d_NUM_PARTICLES);
    hipFree(d_dt);
}