#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call){                                                                    \
    const hipError_t error = call;                                                     \
    if (error != hipSuccess){                                                          \
        printf("Error#: %s:%d \n", __FILE__, __LINE__);                                    \
        printf("\t code:%d, reason: %s\n", error, hipGetErrorString(error));              \
        exit(1);                                                                        \
    }                                                                                   \
}   

void initialInt(int *ip, int size){
    for(int i=0; i<size; i++){
        ip[i] = i;
    }
}

void printMatrix(int *C, const int nx, const int ny){
    int *ic = C;
    printf("\n Matrix: (%d.%d) \n", nx, ny);
    for(int iy=0; iy<ny; iy++){
        for(int ix=0; ix<nx; ix++){
            printf("%3d", ic[ix]);
        }
        ic += nx;
        printf("\n");
    }
    printf("\n");
}

__global__ void printThreadIndex(int *A, const int nx, const int ny){
    int ix = threadIdx.x + blockIdx.x * blockDim.x; 
    int iy = threadIdx.y + blockIdx.y * blockDim.y;    

    unsigned int idx = iy * nx + ix;    
    printf("Thread_id (%d,%d) Block_id (%d,%d) Coordinate (%d,%d) " 
        "global index %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x,
        blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char **argv){
    printf("%s Starting ... \n", argv[0]);

    // Get device Info
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // Set Matrix Dim 
    int nx = 8;
    int ny = 6;
    int nxy = nx*ny;
    int nBytes = nxy * sizeof(float);

    // Malloc host mem
    int *h_A;
    h_A = (int *)malloc(nBytes);

    // Initialise host matrix with int
    initialInt(h_A, nxy);
    printMatrix(h_A, nx, ny);

    // Malloc device memory
    int *d_MatA;
    hipMalloc((void **)&d_MatA, nBytes);

    // Transfer data from host to data
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);

    // Setup execution config 
    dim3 block(4,2);
    dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y );

    // Invoke Kernal
    printThreadIndex <<<grid, block >>>(d_MatA, nx, ny);
    hipDeviceSynchronize();

    // Free host and device Memory
    hipFree(d_MatA);
    free(h_A);

    // Reset device
    hipDeviceReset();

    return 0;

}