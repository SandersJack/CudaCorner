#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void checkIndex(void){
    printf("ThreadIdx: (%d, %d, %d) BlockIdx: (%d, %d, %d) BlockDim: (%d, %d, %d) GridDim (%d, %d, %d) \n", 
            threadIdx.x, threadIdx.y, threadIdx.z,
            blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
            gridDim.x,gridDim.y,gridDim.z);
}

int main(int argc, char **argv){
    // Define total data element
    int nElem = 6;

    // Define grid and block stucture
    dim3 block(3);
    dim3 grid(( nElem + block.x-1) /  block.x);

    // Check grid and block dimension from host side
    printf("grid.x %d grid.y %d grid.z %d\n",grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n",block.x, block.y, block.z);

    checkIndex <<<grid, block>>>();

    // Reset device
    hipDeviceReset();

    return 0;

}